#include "hip/hip_runtime.h"
// https://www.youtube.com/watch?v=jhmgti7OKlQ&t=0s

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
extern "C" {
#include "ppmFile.h"
}

#define TILE_WIDTH 32

__global__ void edgeDetectionTiled(int *d_width, int *d_height, unsigned char *d_input, unsigned char *d_output){

    // allocate shared memory
    __shared__ unsigned char sharedA[TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    if(row >= (*d_height) || col >= (*d_width))
        return;

    // load shared memory
    int num_tiles = ((*d_width)+TILE_WIDTH - 1)/ TILE_WIDTH;
    for(int m = 0; m < num_tiles; m++){
        sharedA[tx] = d_input[0]; // what section do I want to copy?
    }
    // offset of the shared memory as well

    __syncthreads(); // barrier to wait for other threads before calculating

    int offset = 0, currT = 0, t2 = 0, t3 = 0, t4 = 0, t5 = 0;

    if(row > 0 && row < (*d_height) - 1 && col > 0 && col < (*d_width) - 1){
        offset = (row * (*d_width) + col) * 3;
        currT = (d_input[offset] + d_input[offset + 1] + d_input[offset + 2]);

        offset = (row * (*d_width) + (col+1)) * 3;
        t2 = (d_input[offset] + d_input[offset + 1] + d_input[offset + 2]);

        offset = (row * (*d_width) + (col-1)) * 3;
        t3 = (d_input[offset] + d_input[offset + 1] + d_input[offset + 2]);

        offset = ((row+1) * (*d_width) + col) * 3;
        t4 = (d_input[offset] + d_input[offset + 1] + d_input[offset + 2]);

        offset = ((row-1) * (*d_width) + col) * 3;
        t5 = d_input[offset] +  d_input[offset + 1] +  d_input[offset + 2];
    }

    __syncthreads(); // barrier to wait for other threads before replacing output

    offset = (row * (*d_width) + col) * 3; // curr offset
    if (abs(currT - t2) > 100 || abs(currT - t3) > 100 || abs(currT - t4) > 100 || abs(currT - t5) > 100){
        d_output[offset] = 255;
        d_output[offset + 1] = 255;
        d_output[offset + 2] = 255;
    }
    else{
        d_output[offset] = 0;
        d_output[offset + 1] = 0;
        d_output[offset + 2] = 0;
    }
}

int main (int argc, char *argv[]){
    double time_DTH, time_allocateHTD, time_kernel;
    clock_t begin_allocateHTD, end_allocateHTD, begin_kernel, end_kernel, begin_DTH, end_DTH;

    // Host variables (CPU)
    int width, height; //width & heigth for the image

    Image *inImage, *outImage; // ppmFile defined Image Struct
    unsigned char *data; // data of input image

    // Device variable (GPU)
    unsigned char *d_input, *d_output; // input image data
    int *d_width, *d_height; // width & height for the kernel

    if(argc != 3){
        printf("Incorrect number of input arguments. Include an input and output file, in that order.\n");
        return 0;
    }

    // Initializing values
    inImage = ImageRead(argv[1]);
    width = inImage->width;
    height = inImage->height;
    data = inImage->data;

    // Print the values of the images
    printf("Detecting edges on image: %s, with width: %d & height: %d\n", argv[1], width, height);

    // Grids based on size of the block
    dim3 blockD(TILE_WIDTH,TILE_WIDTH);
    dim3 gridD((width + blockD.x - 1)/blockD.x, (height + blockD.y - 1)/blockD.y);

    // Size of image pixels; 3 is the number of channels for RGB
    int image_size = width * height * 3;

    begin_allocateHTD = clock();

    // Allocate memory for GPU
    hipMalloc((void**)&d_input, sizeof(unsigned char*) * image_size);
    hipMalloc((void**)&d_output, sizeof(unsigned char*) * image_size);
    hipMalloc((void**)&d_height, sizeof(int*));
    hipMalloc((void**)&d_width, sizeof(int*));

    // Copy values from Host(CPU) to Device(GPU)
    hipMemcpy(d_input, data, image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);

    end_allocateHTD = clock();

    begin_kernel = clock();

    // Call edgeDetection() kernel on GPU.
    edgeDetectionTiled<<<gridD, blockD>>>(d_width, d_height, d_input, d_output);

    end_kernel = clock();

    // Create new image and clear the image to copy new image from the device.
    outImage = ImageCreate(width,height);
    ImageClear(outImage, 255, 255, 255);


    begin_DTH = clock();
    hipDeviceSynchronize();

    // Copy new (edgeDetected) values from Device(GPU) to Host(CPU)
    hipMemcpy(outImage->data, d_output, image_size, hipMemcpyDeviceToHost);

    end_DTH = clock();

    // Write the new image onto the given file name
    ImageWrite(outImage, argv[2]);

    time_allocateHTD = (double)(end_allocateHTD-begin_allocateHTD) / CLOCKS_PER_SEC;
    printf("Allocation and Host to Device Time: %e s\n", time_allocateHTD);

    time_kernel = (double)(end_kernel-begin_kernel) / CLOCKS_PER_SEC;
    printf("Kernel Time: %e s\n", time_kernel);

    time_DTH = (double)(end_DTH-begin_DTH) / CLOCKS_PER_SEC;
    printf("Device to Host Time: %e s\n", time_DTH);

    printf("Total Time : %e s\n",time_allocateHTD + time_kernel + time_DTH);

    // Free memory
    free(inImage->data);
    free(outImage->data);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_width);
    hipFree(d_height);

    return 0;
}
